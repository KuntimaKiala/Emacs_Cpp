#include "hip/hip_runtime.h"
#include <stdio.h>
#include "/home/kuntima/workspace/github/Emacs_Cpp/constant.h"
#include <string>
#define N 3


__device__ void func(float val){
  val = val/2 ;
  printf("pi/2 = %.9f\n",val) ;
}

__global__ void kernel(const float* float_ptr, const float value){
  
  float_ptr = &value ;
  func(*float_ptr) ;
}

int main(void) {
  
  float * pi ;
  hipMalloc(&pi , sizeof(float)) ;
  kernel<<<1,1>>>(pi, constants::pi);
  hipFree(pi);
  return 0 ;
}

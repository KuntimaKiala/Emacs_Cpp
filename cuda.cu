#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "/home/kuntima/workspace/github/Emacs_Cpp/constant.h"
#define N 3


__device__ void func(float val){
  val = val + val ;
  printf("%f\n",val) ;
}

__global__ void kernel(const float* pt, const float value){
  
  pt = &value ;
  func(*pt) ;
}

int main(void) {
  
  
  float * pi ;
  hipMalloc((void **)&pi , sizeof(float)) ;
  
  kernel<<<1,1>>>(pi, constants::pi);
  hipFree(pi);
  return 0 ;
}

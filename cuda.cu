#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "constant.h"
#define N 3


__device__ void func(const float val){
printf("%f\n",val) ;


}

__global__ void kernel(const float* pt, const float value){
  
  pt = &value ;
  func(*pt) ;
  
}

int main(void) {
  
    
  float * pi ;
  hipMalloc((void **)&pi , sizeof(float)) ;
  kernel<<<1,1>>>(pi, constants::pi);
  hipFree(pi);
  return 0 ;
}
